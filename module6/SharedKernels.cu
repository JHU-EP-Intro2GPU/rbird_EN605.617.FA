#include "hip/hip_runtime.h"
#include "KernelFunctionDefinitions.h"

__global__ void sharedMemAdd(int* output, const int* input1, const int* input2, const size_t count)
{
    extern __shared__ int sharedMem[];

    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // load both values into sharedMem
    sharedMem[blockIdx.x] = input1[tid];
    sharedMem[blockIdx.x + blockDim.x] = input2[tid];

    int result = sharedMem[blockIdx.x] + sharedMem[blockIdx.x + blockDim.x];
    output[tid] = result;
}

__global__ void sharedMemSub(int* output, const int* input1, const int* input2, const size_t count)
{
    extern __shared__ int sharedMem[];

    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // load both values into sharedMem
    sharedMem[blockIdx.x] = input1[tid];
    sharedMem[blockIdx.x + blockDim.x] = input2[tid];

    int result = sharedMem[blockIdx.x] - sharedMem[blockIdx.x + blockDim.x];
    output[tid] = result;
}

__global__ void sharedMemMult(int* output, const int* input1, const int* input2, const size_t count)
{
    extern __shared__ int sharedMem[];

    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // load both values into sharedMem
    sharedMem[blockIdx.x] = input1[tid];
    sharedMem[blockIdx.x + blockDim.x] = input2[tid];

    int result = sharedMem[blockIdx.x] * sharedMem[blockIdx.x + blockDim.x];
    output[tid] = result;
}

__global__ void sharedMemMod(int* output, const int* input1, const int* input2, const size_t count)
{
    extern __shared__ int sharedMem[];

    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // load both values into sharedMem
    sharedMem[blockIdx.x] = input1[tid];
    sharedMem[blockIdx.x + blockDim.x] = input2[tid];

    int result = sharedMem[blockIdx.x] % sharedMem[blockIdx.x + blockDim.x];
    output[tid] = result;
}
