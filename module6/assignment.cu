


#include "hip/hip_runtime.h"
#include ""

#include "assignment.h"
#include "KernelFunctionDefinitions.h"

#include <cstring>
#include <cstdlib>
#include <memory>

#include <stdio.h>

int arraySize = 512;
int blockSize = 32;

DeviceMemory<int> d_source1, d_source2, d_output;

std::unique_ptr<int[]> host_source1, host_source2, host_output;


enum TestKernelType {
    GLOBAL_MEM_ADD, GLOBAL_MEM_SUB, GLOBAL_MEM_MULT, GLOBAL_MEM_MOD,
    SHARED_MEM_ADD, SHARED_MEM_SUB, SHARED_MEM_MULT, SHARED_MEM_MOD,
    REGISTER_MEM_ADD, REGISTER_MEM_SUB, REGISTER_MEM_MULT, REGISTER_MEM_MOD,
    REGISTER_MEM_2_ADD, REGISTER_MEM_2_SUB, REGISTER_MEM_2_MULT, REGISTER_MEM_2_MOD,
    REGISTER_MEM_4_ADD, REGISTER_MEM_4_SUB, REGISTER_MEM_4_MULT, REGISTER_MEM_4_MOD,
    REGISTER_MEM_8_ADD, REGISTER_MEM_8_SUB, REGISTER_MEM_8_MULT, REGISTER_MEM_8_MOD,
};

enum MathOperation {
    ADD, SUB, MULT, MOD
};

char OpToChar(MathOperation operation) {
    switch (operation) {
    case MathOperation::ADD:
        return '+';
    case MathOperation::SUB:
        return '-';
    case MathOperation::MULT:
        return '*';
    case MathOperation::MOD:
        return '%';
    }
}

void populateTestData() {
    d_output.allocate(arraySize);
    d_source1.allocate(arraySize);
    d_source2.allocate(arraySize);

    host_output.reset(new int[arraySize]);
    host_source1.reset(new int[arraySize]);
    host_source2.reset(new int[arraySize]);

    for (int i = 0; i < arraySize; i++) {
        host_source1[i] = rand() % 1000;
        host_source2[i] = rand() % 1000;
    }

    hipMemcpy(d_source1.ptr(), host_source1.get(), arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_source2.ptr(), host_source2.get(), arraySize * sizeof(int), hipMemcpyHostToDevice);
}

void validateCorrectness(MathOperation operation) {
    hipMemcpy(host_output.get(), d_output.ptr(), arraySize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; i++) {
        int expectedAnswer;
        switch (operation) {
        case MathOperation::ADD:
            expectedAnswer = host_source1[i] + host_source2[i];
            break;
        case MathOperation::SUB:
            expectedAnswer = host_source1[i] - host_source2[i];
            break;
        case MathOperation::MULT:
            expectedAnswer = host_source1[i] * host_source2[i];
            break;
        case MathOperation::MOD:
            expectedAnswer = host_source2[i] == 0? -1 : host_source1[i] % host_source2[i];
            break;
        }

        // DEBUG
//        printf("%d: %4d %c %4d = %4d, got: %4d\n", i, host_source1[i], OpToChar(operation), host_source2[i], expectedAnswer, host_output[i]);

        if (host_output[i] != expectedAnswer) {
            printf("%d: ERROR! %4d %c %4d = %4d, got: %4d\n", i, host_source1[i], OpToChar(operation), host_source2[i], expectedAnswer, host_output[i]);
        }
    }
}

void resetOutputBufferData() {
    // clear output buffers
    gpuErrchk(hipMemset(d_output.ptr(), 0, arraySize * sizeof(int)));
    memset(host_output.get(), 0, arraySize * sizeof(int));
}

void testKernelRun(TestKernelType kernelType, const char* description, MathOperation operation) {
    int numBlocks = (arraySize + blockSize - 1) / blockSize;

    // each thread in a block stores two values in shared memory
    int sharedMemoryBytes = blockSize * sizeof(int) * 2;

    {
        TimeCodeBlock kernelRunMeasurement(description);

        switch (kernelType)
        {
        // Global memory tests
        case GLOBAL_MEM_ADD:
            globalMemAdd<<< numBlocks, blockSize>>>(d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case GLOBAL_MEM_SUB:
            globalMemSub <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case GLOBAL_MEM_MULT:
            globalMemMult <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case GLOBAL_MEM_MOD:
            globalMemMod <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;

        // Shared memory tests
        case SHARED_MEM_ADD:
            sharedMemAdd <<< numBlocks, blockSize, sharedMemoryBytes >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case SHARED_MEM_SUB:
            sharedMemSub <<< numBlocks, blockSize, sharedMemoryBytes >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case SHARED_MEM_MULT:
            sharedMemMult <<< numBlocks, blockSize, sharedMemoryBytes >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case SHARED_MEM_MOD:
            sharedMemMod <<< numBlocks, blockSize, sharedMemoryBytes >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;

        // 1 Register memory tests
        case REGISTER_MEM_ADD:
            registerMemAdd <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_SUB:
            registerMemSub <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_MULT:
            registerMemMult <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_MOD:
            registerMemMod <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;

        // 2 Register memory tests
        case REGISTER_MEM_2_ADD:
            numBlocks = ((arraySize / 2) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemAdd_2 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_2_SUB:
            numBlocks = ((arraySize / 2) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemSub_2 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_2_MULT:
            numBlocks = ((arraySize / 2) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMult_2 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_2_MOD:
            numBlocks = ((arraySize / 2) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMod_2 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;

            // 4 Register memory tests
        case REGISTER_MEM_4_ADD:
            numBlocks = ((arraySize / 4) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemAdd_4 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_4_SUB:
            numBlocks = ((arraySize / 4) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemSub_4 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_4_MULT:
            numBlocks = ((arraySize / 4) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMult_4 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_4_MOD:
            numBlocks = ((arraySize / 4) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMod_4 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;

            // 8 Register memory tests
        case REGISTER_MEM_8_ADD:
            numBlocks = ((arraySize / 8) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemAdd_8 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_8_SUB:
            numBlocks = ((arraySize / 8) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemSub_8 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_8_MULT:
            numBlocks = ((arraySize / 8) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMult_8 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;
        case REGISTER_MEM_8_MOD:
            numBlocks = ((arraySize / 8) + blockSize - 1) / blockSize;
            numBlocks = (numBlocks == 0) ? 1 : numBlocks;
            registerMemMod_8 <<< numBlocks, blockSize >>> (d_output.ptr(), d_source1.ptr(), d_source2.ptr(), arraySize);
            break;


        default:
            break;
        }

        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    validateCorrectness(operation);
    resetOutputBufferData();
}

void testKernels() {
    printf("Arraysize: %d Blocksize: %d\n", arraySize, blockSize);
    populateTestData();

    resetOutputBufferData();

    printf("--------------- GLOBAL MEMORY TESTS -------------------------\n");

    testKernelRun(TestKernelType::GLOBAL_MEM_ADD, "Global Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::GLOBAL_MEM_SUB, "Global Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::GLOBAL_MEM_MULT, "Global Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::GLOBAL_MEM_MOD, "Global Memory Mod Kernel", MathOperation::MOD);


    printf("\n--------------- SHARED MEMORY TESTS -------------------------\n");
    testKernelRun(TestKernelType::SHARED_MEM_ADD, "Shared Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::SHARED_MEM_SUB, "Shared Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::SHARED_MEM_MULT, "Shared Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::SHARED_MEM_MOD, "Shared Memory Mod Kernel", MathOperation::MOD);


    printf("\n--------------- REGISTER (1) MEMORY TESTS -------------------------\n");
    testKernelRun(TestKernelType::REGISTER_MEM_ADD, "Register Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::REGISTER_MEM_SUB, "Register Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::REGISTER_MEM_MULT, "Register Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::REGISTER_MEM_MOD, "Register Memory Mod Kernel", MathOperation::MOD);

    printf("\n--------------- REGISTER (2) MEMORY TESTS -------------------------\n");
    testKernelRun(TestKernelType::REGISTER_MEM_2_ADD, "Register (2) Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::REGISTER_MEM_2_SUB, "Register (2) Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::REGISTER_MEM_2_MULT, "Register (2) Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::REGISTER_MEM_2_MOD, "Register (2) Memory Mod Kernel", MathOperation::MOD);

    // TODO: 4 registers, maybe 8
    printf("\n--------------- REGISTER (4) MEMORY TESTS -------------------------\n");
    testKernelRun(TestKernelType::REGISTER_MEM_4_ADD, "Register (4) Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::REGISTER_MEM_4_SUB, "Register (4) Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::REGISTER_MEM_4_MULT, "Register (4) Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::REGISTER_MEM_4_MOD, "Register (4) Memory Mod Kernel", MathOperation::MOD);

    printf("\n--------------- REGISTER (8) MEMORY TESTS -------------------------\n");
    testKernelRun(TestKernelType::REGISTER_MEM_8_ADD, "Register (8) Memory Add Kernel", MathOperation::ADD);
    testKernelRun(TestKernelType::REGISTER_MEM_8_SUB, "Register (8) Memory Sub Kernel", MathOperation::SUB);
    testKernelRun(TestKernelType::REGISTER_MEM_8_MULT, "Register (8) Memory Mult Kernel", MathOperation::MULT);
    testKernelRun(TestKernelType::REGISTER_MEM_8_MOD, "Register (8) Memory Mod Kernel", MathOperation::MOD);
}


int main(int argc, char* argv[])
{
    for (int i = 0; i < argc; i++) {
        const char* arg = argv[i];
        if (strcmp(arg, "--elements") == 0) {
            i++;
            arraySize = atoi(argv[i]);
        }
        else if (strcmp(arg, "--blocksize") == 0) {
            i++;
            blockSize = atoi(argv[i]);
        }
    }

    testKernels();

    return 0;
}

