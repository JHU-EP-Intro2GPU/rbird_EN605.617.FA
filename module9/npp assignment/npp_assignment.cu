/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#  pragma warning(disable:4819)
#endif

#include <ImagesCPU.h>
#include <ImagesNPP.h>
#include <ImageIO.h>
#include <Exceptions.h>

#include <string.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <npp.h>

#include <helper_string.h>
#include <hip/hip_runtime_api.h>

#include "assignment.h"

    void
    saveImage(const std::string &rFileName, const npp::ImageCPU_8u_C3 &rImage)
    {
        // create the result image storage using FreeImage so we can easily
        // save
        FIBITMAP *pResultBitmap = FreeImage_Allocate(rImage.width(), rImage.height(), 24 /* bits per pixel, 8 bits, 3 channels */);
        NPP_ASSERT_NOT_NULL(pResultBitmap);
        unsigned int nDstPitch   = FreeImage_GetPitch(pResultBitmap);
        Npp8u *pDstLine = FreeImage_GetBits(pResultBitmap) + nDstPitch * (rImage.height()-1);
        const Npp8u *pSrcLine = rImage.data();
        unsigned int nSrcPitch = rImage.pitch();

        for (size_t iLine = 0; iLine < rImage.height(); ++iLine)
        {
            memcpy(pDstLine, pSrcLine, rImage.width() * (3 * sizeof(Npp8u)));
            pSrcLine += nSrcPitch;
            pDstLine -= nDstPitch;
        }

        // now save the result image
        bool bSuccess;
        bSuccess = FreeImage_Save(FIF_PGM, pResultBitmap, rFileName.c_str(), 0) == TRUE;
        NPP_ASSERT_MSG(bSuccess, "Failed to save result image.");
    }

inline int cudaDeviceInit(int argc, const char **argv)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        std::cerr << "CUDA error: no devices supporting CUDA." << std::endl;
        exit(EXIT_FAILURE);
    }

    int dev = findCudaDevice(argc, argv);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    std::cerr << "hipSetDevice GPU" << dev << " = " << deviceProp.name << std::endl;

    checkCudaErrors(hipSetDevice(dev));

    return dev;
}

bool printfNPPinfo(int argc, char *argv[])
{
    const NppLibraryVersion *libVer   = nppGetLibVersion();

    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  CUDA Driver  Version: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);

    // Min spec is SM 1.0 devices
    bool bVal = checkCudaCapabilities(1, 0);
    return bVal;
}

int main(int argc, char *argv[])
{
    printf("%s Starting...\n\n", argv[0]);

    try
    {
        std::string sFilename;
        char *filePath;

        cudaDeviceInit(argc, (const char **)argv);

        if (printfNPPinfo(argc, argv) == false)
        {
            exit(EXIT_SUCCESS);
        }

        if (checkCmdLineFlag(argc, (const char **)argv, "input"))
        {
            getCmdLineArgumentString(argc, (const char **)argv, "input", &filePath);
        }
        else
        {
            filePath = sdkFindFilePath("Lena.pgm", argv[0]);
        }

        if (filePath)
        {
            sFilename = filePath;
        }
        else
        {
            sFilename = "Lena.pgm";
        }

        // if we specify the filename at the command line, then we only test sFilename[0].
        int file_errors = 0;
        std::ifstream infile(sFilename.data(), std::ifstream::in);

        if (infile.good())
        {
            std::cout << "boxFilterNPP opened: <" << sFilename.data() << "> successfully!" << std::endl;
            file_errors = 0;
            infile.close();
        }
        else
        {
            std::cout << "boxFilterNPP unable to open: <" << sFilename.data() << ">" << std::endl;
            file_errors++;
            infile.close();
        }

        if (file_errors > 0)
        {
            exit(EXIT_FAILURE);
        }

        std::string sResultFilename = sFilename;

        std::string::size_type dot = sResultFilename.rfind('.');

        if (dot != std::string::npos)
        {
            sResultFilename = sResultFilename.substr(0, dot);
        }

        sResultFilename += "_debayerFilter.pgm";

        if (checkCmdLineFlag(argc, (const char **)argv, "output"))
        {
            char *outputFilePath;
            getCmdLineArgumentString(argc, (const char **)argv, "output", &outputFilePath);
            sResultFilename = outputFilePath;
        }
    
        TimeCodeBlockCuda processRuntime("Entire process");

        // declare a host image object for an 8-bit grayscale image
        npp::ImageCPU_8u_C1 oHostSrc;
        // load gray-scale image from disk
        npp::loadImage(sFilename, oHostSrc);

        // declare a device image and copy construct from the host image,
        // i.e. upload host to device
        npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);

        // create struct with box-filter mask size

        NppiSize oSrcSize;
        oSrcSize.width = (int)oDeviceSrc.width();
        oSrcSize.height = (int)oDeviceSrc.height();

        // create struct with ROI size
        NppiRect oSizeROI;
        oSizeROI.x = 0;
        oSizeROI.y = 0;
    
    
        oSizeROI.width = (int)oDeviceSrc.width();
        oSizeROI.height = (int)oDeviceSrc.height();

        // allocate device image of appropriately reduced size
        npp::ImageNPP_8u_C3 oDeviceDst(oDeviceSrc.width(), oDeviceSrc.height());

        // run debayer color filter

        //eInterpolation	MUST be NPPI_INTER_UNDEFINED
        // https://docs.nvidia.com/cuda/npp/group__image__color__debayer.html
        std::printf("src pitch: %d destPitch: %d\n", oDeviceSrc.pitch(), oDeviceDst.pitch());
        std::printf("src size height: %d width: %d\n", oSrcSize.height, oSrcSize.width);
        std::printf("dst size height: %d width: %d\n", oDeviceDst.height(), oDeviceDst.width());
        std::printf("roi height: %d width: %d\n", oSizeROI.height, oSizeROI.width);
        
        {
            TimeCodeBlockCuda conversion("Grayscale conversion");
            NPP_CHECK_NPP(
                nppiCFAToRGB_8u_C1C3R(
                    oDeviceSrc.data(), oDeviceSrc.pitch(), oSrcSize, oSizeROI,
                    oDeviceDst.data(), oDeviceDst.pitch(), NPPI_BAYER_GRBG, NPPI_INTER_UNDEFINED
                )
            );
        }

        // declare a host image for the result
        npp::ImageCPU_8u_C3 oHostDst(oDeviceDst.size());
        // and copy the device result data into it
        oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());

        saveImage(sResultFilename, oHostDst);
        std::cout << "Saved image: " << sResultFilename << std::endl;

        nppiFree(oDeviceSrc.data());
        nppiFree(oDeviceDst.data());
    }
    catch (npp::Exception &rException)
    {
        std::cerr << "Program error! The following exception occurred: \n";
        std::cerr << rException << std::endl;
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
    }
    catch (...)
    {
        std::cerr << "Program error! An unknow type of exception occurred. \n";
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
        return -1;
    }

    return 0;
}
