#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void what_is_my_id_2d_A(
	unsigned int* const block_x,
	unsigned int* const block_y,
	unsigned int* const thread,
	unsigned int* const calc_thread,
	unsigned int* const x_thread,
	unsigned int* const y_thread,
	unsigned int* const grid_dimx,
	unsigned int* const block_dimx,
	unsigned int* const grid_dimy,
	unsigned int* const block_dimy)
{
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

	block_x[thread_idx] = blockIdx.x;
	block_y[thread_idx] = blockIdx.y;
	thread[thread_idx] = threadIdx.x;
	calc_thread[thread_idx] = thread_idx;
	x_thread[thread_idx] = idx;
	y_thread[thread_idx] = idy;
	grid_dimx[thread_idx] = gridDim.x;
	block_dimx[thread_idx] = blockDim.x;
	grid_dimy[thread_idx] = gridDim.y;
	block_dimy[thread_idx] = blockDim.y;
}

#ifndef ARRAY_SIZE_X
#define ARRAY_SIZE_X 32
#endif // !ARRAY_SIZE_X

#ifndef ARRAY_SIZE_Y
#define ARRAY_SIZE_Y 16
#endif // !ARRAY_SIZE_Y


#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))

/* Declare statically six arrays of ARRAY_SIZE each */
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

int main(void)
{
	/* Total thread count = 32 * 4 = 128 */
	const dim3 threads_rect(32, 4);
	const dim3 blocks_rect(1, 4);

	/* Total thread count = 16 * 8 = 128 */
	const dim3 threads_square(16, 8); /* 16 * 8 */
	const dim3 blocks_square(2, 2);

	/* Total thread count = 8 * 16 = 128 */
	const dim3 reverse_threads_square(8, 16); /* 8 * 16 */
	const dim3 reverse_blocks_square(2, 2);

	/* Total thread count = 4 * 32 = 128 */
	const dim3 reverse_threads_rect(4, 32); /* 4 * 32 */
	const dim3 reverse_blocks_rect(4, 1);

	/* Total thread count = 4 * 2 = 8 */
	const dim3 few_threads_rect(4, 2); /* 4 * 2 */
	const dim3 many_blocks_rect(8, 2);


	/* Total thread count = 4 * 2 = 8 */
	const dim3 many_threads(1, 512); /* 4 * 2 */
	const dim3 one_blocks_rect(1, 1);


	/* Needed to wait for a character at exit */
	char ch;

	/* Declare statically six arrays of ARRAY_SIZE each */
	unsigned int* gpu_block_x;
	unsigned int* gpu_block_y;
	unsigned int* gpu_thread;
	unsigned int* gpu_warp;
	unsigned int* gpu_calc_thread;
	unsigned int* gpu_xthread;
	unsigned int* gpu_ythread;
	unsigned int* gpu_grid_dimx;
	unsigned int* gpu_block_dimx;
	unsigned int* gpu_grid_dimy;
	unsigned int* gpu_block_dimy;

	/* Allocate arrays on the GPU */
	hipMalloc((void**)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_warp, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

	for (int kernel = 0; kernel < 6; kernel++)
	{
		dim3 threads_dim;
		dim3 blocks_dim;
		switch (kernel)
		{
		case 0:
		{
			blocks_dim = blocks_rect;
			threads_dim = threads_rect;
			/* Execute our kernel */
			what_is_my_id_2d_A << <blocks_rect, threads_rect >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;

		case 1:
		{
			blocks_dim = blocks_square;
			threads_dim = threads_square;

			/* Execute our kernel */
			what_is_my_id_2d_A << <blocks_square, threads_square >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;
		case 2:
		{
			blocks_dim = reverse_blocks_square;
			threads_dim = reverse_threads_square;

			/* Execute our kernel */
			what_is_my_id_2d_A << <reverse_blocks_square, reverse_threads_square >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;
		case 3:
		{
			blocks_dim = reverse_blocks_square;
			threads_dim = reverse_threads_rect;

			/* Execute our kernel */
			what_is_my_id_2d_A << <reverse_blocks_square, reverse_threads_rect >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;
		case 4:
		{
			blocks_dim = many_blocks_rect;
			threads_dim = few_threads_rect;

			/* Execute our kernel */
			what_is_my_id_2d_A << <many_blocks_rect, few_threads_rect >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;
		case 5:
		{
			blocks_dim = many_threads;
			threads_dim = one_blocks_rect;

			/* Execute our kernel */
			what_is_my_id_2d_A << <many_threads, one_blocks_rect >> > (gpu_block_x, gpu_block_y,
				gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
				gpu_grid_dimy, gpu_block_dimy);
		} break;


		default: exit(1); break;
		}

		/* Copy back the gpu results to the CPU */
		hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

		printf("\nKernel %d\n", kernel);
		printf("Block Dimensions: x= %2d, y=%2d, z=%2d\n", blocks_dim.x, blocks_dim.y, blocks_dim.z);
		printf("Thread Dimensions: x= %2d, y=%2d, z=%2d\n", threads_dim.x, threads_dim.y, threads_dim.z);
		/* Iterate through the arrays and print */
		for (int y = 0; y < ARRAY_SIZE_Y; y++)
		{
			for (int x = 0; x < ARRAY_SIZE_X; x++)
			{
				printf("CT: %3u BKX: %2u BKY: %2u TID: %3u YTID: %2u XTID: %2u GDX: %1u BDX: %1u GDY: %1u BDY: %1u\n",
					cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x], cpu_thread[y][x], cpu_ythread[y][x],
					cpu_xthread[y][x], cpu_grid_dimx[y][x], cpu_block_dimx[y][x], cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);

			}
		}


	}

	/* Free the arrays on the GPU as now we're done with them */
	hipFree(gpu_block_x);
	hipFree(gpu_block_y);
	hipFree(gpu_thread);
	hipFree(gpu_warp);
	hipFree(gpu_calc_thread);
	hipFree(gpu_xthread);
	hipFree(gpu_ythread);
	hipFree(gpu_grid_dimy);
	hipFree(gpu_block_dimy);
}
