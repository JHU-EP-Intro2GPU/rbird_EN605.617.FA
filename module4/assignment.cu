#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "assignment.h"

#include <stdio.h>
#include <chrono>
#include <cstring>
#include <cstdlib>
#include <iostream>
#include <time.h>

void populateTestData(int threadCount, int blocksize);
void printVector(const HostMemory<int>&, size_t countToUse);
void performCalculations(int blocksize);
void runVerification();

#pragma region Cuda Math Kernels
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void subtractKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] - b[i];
}

__global__ void multiplyKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] * b[i];
}

__global__ void modulusKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] % b[i];
}


__host__ __device__ char convert(const char c, const int offset) {
    int minChar, maxChar;
    if ('a' <= c && c <= 'z') {
        minChar = 'a';
        maxChar = 'z';
    }
    else if ('A' <= c && c <= 'Z') {
        minChar = 'A';
        maxChar = 'Z';
    }
    else {
        // assume space character
        minChar = ' ';
        maxChar = ' ';
    }

    // remap the new offest to be within the min/max boundary.
    // Do this by using the mod operator. Values that are negative
    // need to be positive.

    // Example: 'b' with offset -3 -> 'y'
    //     characterOffest = 'b' - 'a' = 98 - 97 = 1
    //     characterOffest = 1 + (-3) = -2
    //     increment = -2 + 26 = 24
    //     modulus = 24 % 26 = 24
    //     newCharacter = 'a' + 24 = 97 + 24 = 121 = 'y'

    // calculate distance from lowest ascii
    int characterOffest = c - minChar;
    characterOffest += offset;

    int alphabetCount = (maxChar - minChar) + 1;

    // Use the mod operator to keep the offset within the min/max character range
    characterOffest = (characterOffest + alphabetCount) % alphabetCount;

    return (char)((int)minChar) + characterOffest;
}

__global__ void caesarCypher(char* output, const char* input, int bufferSize, const int offset)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < bufferSize) {
        output[i] = convert(input[i], offset);
    }
}

#pragma endregion

enum KernalToRun {
    RunAddKernel, RunSubtractKernel, RunMultiplyKernel, RunModulusKernel
};

HostMemory<int> firstSourceArray;
HostMemory<int> secondSourceArray;

HostMemory<int> addResults;
HostMemory<int> subtractResults;
HostMemory<int> multiplyResults;
HostMemory<int> modulusResults;

bool printDebug = false;
bool verifyCorrectness = false;

bool usePinnedMemory = false;

const char* messageToDecode = "abcdefghijklmnopqrstuvwxyz ABCDEFGHIJKLMNOPQRSTUVWXYZ";
int cipherOffset = -3;

// run the previous Module 3 code on pinned/pageable memory
void performMathProgram(int totalThreadCount, int blockSize);
void performCaesarCipher(const char* value, int cipherOffset, int blockSize);

int main(int argc, char* argv[])
{
    int totalThreadCount = 20;
    int blockSize = 20; // number of threads per block

    // parse the command line arguments
    for (int i = 0; i < argc; i++) {
        const char* arg = argv[i];
        if (strcmp(arg, "--help") == 0 || strcmp(arg, "-h") == 0) {
            printf("USAGE: ./assignment.exe [num_threads] [blocksize]");
            printf("Optional Arguments:\n");
            printf("--help, -h: Show this message and exit\n\n");
            printf("--blocksize [block_size]: set the block size to use on the cuda kernels. Default to %d\n", blockSize);
            printf("--pinned: use pinned host memory for data transfer\n");
            printf("--pageable: use pageable host memory for data transfer\n");
            printf("--threads [thread_count]: set the thread count (and test array size) to use on the cuda kernels. Default to %d\n", totalThreadCount);
            printf("--debug: print out the data to determine test input and output\n");
            printf("--verify: verify that the results from the gpu are the expected values\n");
            printf("\nCipher arguments:\n");
            printf("--message [message]: the message to run the cipher on. Capital and lower case ascii characters only.");
            printf("--offset [offset]: the offset to use for the cipher.");
            return 0;
        }
        else if (strcmp(arg, "--blocksize") == 0) {
            i++;
            blockSize = atoi(argv[i]);
        }
        else if (strcmp(arg, "--threads") == 0) {
            i++;
            totalThreadCount = atoi(argv[i]);
        }
        else if (strcmp(arg, "--debug") == 0) {
            printDebug = true;
        }
        else if (strcmp(arg, "--verify") == 0) {
            verifyCorrectness = true;
        }
        else if (strcmp(arg, "--pinned") == 0) {
            usePinnedMemory = true;
        }
        else if (strcmp(arg, "--pageable") == 0) {
            usePinnedMemory = false;
        }
        else if (strcmp(arg, "--message") == 0) {
            i++;
            messageToDecode = argv[i];
        }
        else if (strcmp(arg, "--offset") == 0) {
            i++;
            cipherOffset = atoi(argv[i]);
        }
        else if (i == 1) {
            totalThreadCount = atoi(arg);
        }
        else if (i == 2) {
            blockSize = atoi(arg);
        }
    }

    performMathProgram(totalThreadCount, blockSize);

    // debug
//    for (int i = 0; i <= 26; i++) {
//        performCaesarCipher("abcdefghijklmnopqrstuvwxyz ABCDEFGHIJKLMNOPQRSTUVWXYZ", i);
//    }

    performCaesarCipher(messageToDecode, cipherOffset, blockSize);


    // Deallocate resources before calling device reset (causes errors during destruction of static objects)
    firstSourceArray.deallocate();
    secondSourceArray.deallocate();

    addResults.deallocate();
    subtractResults.deallocate();
    multiplyResults.deallocate();
    modulusResults.deallocate();


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    gpuErrchk(hipDeviceReset());

    return 0;
}

void performMathProgram(int totalThreadCount, int blockSize) {
    printf("Using %d total threads with %d threads per block\n", totalThreadCount, blockSize);

    if (usePinnedMemory) {
        printf("Using pinned host memory for program execution.\n");
    }
    else {
        printf("Using pageable host memory for program execution.\n");
    }

    populateTestData(totalThreadCount, blockSize);

    if (printDebug) {
        printf("first array:\n");
        printVector(firstSourceArray, totalThreadCount);
        printf("\n\nsecond array:\n");
        printVector(secondSourceArray, totalThreadCount);
        printf("\n\n");
    }

    performCalculations(blockSize);

    if (printDebug) {
        for (int i = 0; i < totalThreadCount; i++) {
            int a = firstSourceArray.ptr()[i];
            int b = secondSourceArray.ptr()[i];

            printf("%3d: %2d + %d = %3d    ", i, a, b, addResults.ptr()[i]);
            printf("%2d - %d = %3d    ", a, b, subtractResults.ptr()[i]);
            printf("%2d * %d = %3d    ", a, b, multiplyResults.ptr()[i]);
            printf("%2d %% %d = %3d    ", a, b, modulusResults.ptr()[i]);
            printf("\n");
        }
    }

    if (verifyCorrectness) {
        runVerification();
    }
}


void performCaesarCipher(const char* value, int cipherOffset, int blockSize) {
    size_t len = strlen(value);
    size_t totalBytes = len * sizeof(char);

    int totalThreadCount = len;
    int totalBlocks = (totalThreadCount + blockSize - 1) / blockSize;

    // Debug
//    for (size_t i = 0; i < len; i++) {
//        printf("%c", convert(value[i], cipherOffset));
//    }

//    printf("\n");

    HostMemory<char> encodedCipherText;
    HostMemory<char> decodedCipherText;

    {
        TimeCodeBlock cipherTextHostAllocation("CipherText Host Allocation");
        encodedCipherText.allocate(len, usePinnedMemory);
        decodedCipherText.allocate(len, usePinnedMemory);
    }

    DeviceMemory<char> device_SourceText(len);
    DeviceMemory<char> device_EncodedText(len);
    DeviceMemory<char> device_DecodedText(len);

    // Send cipher to device
    {
        TimeCodeBlock dataTransferToDevice("CipherText Data Transfer from host to device");
        gpuErrchk(hipMemcpy(device_SourceText.ptr(), value, totalBytes, hipMemcpyHostToDevice));
    }

    // Encode and decode
    {
        TimeCodeBlock cipherTextHostAllocation("CipherText encode");

        // TODO: Debug!
        caesarCypher << <totalBlocks, blockSize >> > (device_EncodedText.ptr(), device_SourceText.ptr(), len, cipherOffset);

        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    {
        TimeCodeBlock cipherTextHostAllocation("CipherText decode");

        // run with negative offset to decode
        // TODO: Debug!
        caesarCypher << <totalBlocks, blockSize >> > (device_DecodedText.ptr(), device_EncodedText.ptr(), len, -cipherOffset);

        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    // Read data back from device
    {
        TimeCodeBlock dataTransferToDevice("CipherText Data Transfer from device to host");
        gpuErrchk(hipMemcpy(encodedCipherText.ptr(), device_EncodedText.ptr(), totalBytes, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(decodedCipherText.ptr(), device_DecodedText.ptr(), totalBytes, hipMemcpyDeviceToHost));
    }

    // print values:
    printf("Encoded:\n");
    for (size_t i = 0; i < len; i++) {
        // memory may not have null terminator. Print one character at a time
        printf("%c", encodedCipherText.ptr()[i]);
    }

    printf("\n\nDecoded:\n");
    for (size_t i = 0; i < len; i++) {
        // memory may not have null terminator. Print one character at a time
        printf("%c", decodedCipherText.ptr()[i]);
    }
    printf("\n");

}

// Use helper function tomake performCalculations more readable
void testKernel(KernalToRun runKernel, const int numBlocks, const int blocksize, int* device_destPtr, int* device_Source1, int* device_Source2);

void performCalculations(int blocksize)
{
    hipError_t cudaStatus;

    size_t bufferCount = firstSourceArray.size();
    const size_t totalBytes = bufferCount * sizeof(int);
    const int numBlocks = bufferCount / blocksize;

    // input sources
    TimeCodeBlock* deviceAllocation = new TimeCodeBlock("Device Allocation");
    DeviceMemory<int> dev_firstSource(bufferCount);
    DeviceMemory<int> dev_secondSource(bufferCount);

    // output sources
    DeviceMemory<int> dev_addResults(bufferCount);
    DeviceMemory<int> dev_subtractResults(bufferCount);
    DeviceMemory<int> dev_multiplyResults(bufferCount);
    DeviceMemory<int> dev_modulusResults(bufferCount);

    delete deviceAllocation;
    deviceAllocation = nullptr;

    // Copy input vectors from host memory to GPU buffers.
    {
        TimeCodeBlock dataTransferToDevice("Data Transfer from host to device");
        gpuErrchk(hipMemcpy(dev_firstSource.ptr(), firstSourceArray.ptr(), totalBytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(dev_secondSource.ptr(), secondSourceArray.ptr(), totalBytes, hipMemcpyHostToDevice));
    }

    // Wait for kernels to finish so that we can properly time each kernel run
    gpuErrchk(hipDeviceSynchronize());

    // Launch and time each AddKernel (helper function synchronizes with device and prints out time)
    printf("Running AddKernel\n");
    testKernel(KernalToRun::RunAddKernel, numBlocks, blocksize, dev_addResults.ptr(), dev_firstSource.ptr(), dev_secondSource.ptr());

    printf("Running SubtractKernel\n");
    testKernel(KernalToRun::RunSubtractKernel, numBlocks, blocksize, dev_subtractResults.ptr(), dev_firstSource.ptr(), dev_secondSource.ptr());

    printf("Running MultiplyKernel\n");
    testKernel(KernalToRun::RunMultiplyKernel, numBlocks, blocksize, dev_multiplyResults.ptr(), dev_firstSource.ptr(), dev_secondSource.ptr());

    printf("Running ModulusKernel\n");
    testKernel(KernalToRun::RunModulusKernel, numBlocks, blocksize, dev_modulusResults.ptr(), dev_firstSource.ptr(), dev_secondSource.ptr());


    // Copy output vectors from GPU buffer to host memory.
    {
        TimeCodeBlock dataTransferToDevice("Data Transfer from device to host");
        gpuErrchk(hipMemcpy(addResults.ptr(), dev_addResults.ptr(), totalBytes, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(subtractResults.ptr(), dev_subtractResults.ptr(), totalBytes, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(multiplyResults.ptr(), dev_multiplyResults.ptr(), totalBytes, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(modulusResults.ptr(), dev_modulusResults.ptr(), totalBytes, hipMemcpyDeviceToHost));
    }
}

void testKernel(KernalToRun runKernel, const int numBlocks, const int blocksize, int* device_destPtr, int* device_Source1, int* device_Source2) {
    if (printDebug) {
        printf("\tLaunching %d blocks with %d threads per block.\n", numBlocks, blocksize);
    }

    TimeCodeBlock timeKernelRun("\tKernelRun");

    switch (runKernel)
    {
    case RunAddKernel:
        addKernel << < numBlocks, blocksize >> > (device_destPtr, device_Source1, device_Source2);
        break;
    case RunSubtractKernel:
        subtractKernel << < numBlocks, blocksize >> > (device_destPtr, device_Source1, device_Source2);
        break;
    case RunMultiplyKernel:
        multiplyKernel << < numBlocks, blocksize >> > (device_destPtr, device_Source1, device_Source2);
        break;
    case RunModulusKernel:
        modulusKernel << < numBlocks, blocksize >> > (device_destPtr, device_Source1, device_Source2);
        break;
    default:
        break;
    }

    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
}

void populateTestData(const int threadCount, const int blocksize)
{
    // Reserve enough data so that we don't have any out of bounds
    // memory access (prevent the need to check device array size).
    // We need a number >= threadcount that is evenly divisible by blocksize
    size_t reserveSize = 0;
    const int extraThreads = threadCount % blocksize;
    if (extraThreads == 0)
        reserveSize = threadCount; // blocks fit exactly
    else
        reserveSize = threadCount + (blocksize - extraThreads); // add more threads to be a multiple of 'blocksize'

    {
        TimeCodeBlock hostAllocation("Allocate Host Memory");
        firstSourceArray.allocate(reserveSize, usePinnedMemory);
        secondSourceArray.allocate(reserveSize, usePinnedMemory);

        addResults.allocate(reserveSize, usePinnedMemory);
        subtractResults.allocate(reserveSize, usePinnedMemory);
        multiplyResults.allocate(reserveSize, usePinnedMemory);
        modulusResults.allocate(reserveSize, usePinnedMemory);
    }

    {
        TimeCodeBlock populateHostData("Populate host data");
        // Populate the first array: "the first should contain values from 0 - total number of threads"
        for (int i = 0; i < threadCount; i++)
            firstSourceArray.ptr()[i] = i;

        // the second with random values between 0 and 3
        int maxValueExclusive = 4;

        // randomize the seed
        srand(time(NULL));
        for (int i = 0; i < threadCount; i++)
            secondSourceArray.ptr()[i] = rand() % maxValueExclusive;
    }
}

void printVector(const HostMemory<int>& values, size_t countToUse) {
    printf("[");

    const int* data = values.ptr();
    for (size_t i = 0; i < countToUse; i++) {
        if (i != 0) {
            printf(",");
        }

        printf(" %d", data[i]);
    }

    printf("]");
}

void runVerification() {
    int errorCount = 0;

    for (size_t i = 0; i < firstSourceArray.size(); i++) {
        int a = firstSourceArray.ptr()[i];
        int b = secondSourceArray.ptr()[i];

        if (a + b != addResults.ptr()[i]) {
            printf("ERROR: %d Add is incorrect\n", i);
            errorCount++;
        }

        if (a - b != subtractResults.ptr()[i]) {
            printf("ERROR: %d Subtract is incorrect\n", i);
            errorCount++;
        }

        if (a * b != multiplyResults.ptr()[i]) {
            printf("ERROR: %d Multiply is incorrect\n", i);
            errorCount++;
        }

        if (b == 0) {
            if (modulusResults.ptr()[i] != -1) {
                printf("ERROR: %d Modulus is incorrect\n", i);
                errorCount++;
            }
        }
        else if (a % b != modulusResults.ptr()[i]) {
            printf("ERROR: %d Modulus is incorrect\n", i);
            errorCount++;
        }

    }

    if (errorCount == 0) {
        printf("Verification Success!\n");
    }

}
