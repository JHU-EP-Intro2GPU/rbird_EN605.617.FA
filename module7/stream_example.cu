/*https://cdac.in/index.aspx?id=ev_hpc_gpu-comp-nvidia-cuda-streams#hetr-cuda-prog-cuda-streams*/

#include <stdio.h> 
#include <time.h> 
#include <hip/hip_runtime.h> 

#define sizeOfArray 1024*1024 * 15


constexpr int arraySizePerCall = 1024 * 1024;

static_assert (sizeOfArray% arraySizePerCall == 0, "stream size must divide evenly into the total size");

__global__ void arrayAddition(int *device_a, int *device_b, int *device_result, int iteration)
{

	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	threadId += iteration * arraySizePerCall;

	if (threadId < sizeOfArray) 
        device_result[threadId]= device_a[threadId]+device_b[threadId]; 
} 


/* Check for safe return of all calls to the device */ 

int main ( int argc, char **argv ) 
{ 

  hipDeviceProp_t prop; 
  int *host_a, *host_b, *host_result; 
  int *device_a, *device_b, *device_result; 
  int whichDevice; 

  hipGetDeviceCount( &whichDevice); 
  hipGetDeviceProperties( &prop, whichDevice); 

  hipEvent_t start, stop; 
  float elapsedTime; 

  hipEventCreate( &start ); 
  hipEventCreate( &stop ); 

  hipStream_t stream; 
  hipStreamCreate(&stream); 

  hipMalloc( ( void**)& device_a, sizeOfArray * sizeof ( *device_a ) ); 
  hipMalloc( ( void**)& device_b,sizeOfArray * sizeof ( *device_b ) ); 
  hipMalloc( ( void**)& device_result, sizeOfArray * sizeof ( *device_result ) ); 

  hipHostAlloc((void **)&host_a, sizeOfArray*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_b, sizeOfArray*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&host_result, sizeOfArray*sizeof(int), hipHostMallocDefault);

  for(int index = 0; index < sizeOfArray; index++) 
  { 
   host_a[index] = rand()%10; 
   host_b[index] = rand()%10; 
  } 

  hipEventRecord(start);

  hipMemcpyAsync(device_a, host_a,sizeOfArray * sizeof ( int ), hipMemcpyHostToDevice, stream); 

  hipMemcpyAsync(device_b, host_b, sizeOfArray * sizeof ( int ), hipMemcpyHostToDevice, stream); 

  /*Kernel call*/ 
  int totalIterations = sizeOfArray / arraySizePerCall;
  for (int iteration = 0; iteration < totalIterations; iteration++) {
	  arrayAddition <<<sizeOfArray, 1, 1, stream >>> (device_a, device_b, device_result, iteration);
	  int ptrOffset = iteration * arraySizePerCall;
	  hipMemcpyAsync(host_result + ptrOffset, device_result + ptrOffset, arraySizePerCall * sizeof(int), hipMemcpyDeviceToHost, stream);
  }

  hipStreamSynchronize(stream);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  hipEventElapsedTime(&elapsedTime, start, stop); 

  printf("*********** CDAC - Tech Workshop : hyPACK-2013 \n"); 
  printf("\n Size of array : %d \n", sizeOfArray); 
  printf("\n Time taken: %3.1f ms \n", elapsedTime); 

  for (int i = 0; i < sizeOfArray; i++) {
	  if (host_a[i] + host_b[i] != host_result[i]) {
		  printf("ERROR(%d): %d + %d = %d\n", host_a[i], host_b[i], host_result[i]);
	  }
  }

  hipHostFree(host_a); 
  hipHostFree(host_b); 
  hipHostFree(host_result); 
  hipFree(device_a); 
  hipFree(device_b); 
  hipFree(device_result); 

  return 0; 
}
